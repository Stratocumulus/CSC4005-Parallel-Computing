#include "hip/hip_runtime.h"
#include <cstring>
#include <graphic/graphic.hpp>
#include <imgui_impl_sdl.h>
#include <nbody/body.hpp>

template <typename... Args> void UNUSED(Args &&...args [[maybe_unused]]) {}

__global__ void cudaCheckBodies(double *x, double *y, double *vx, 
                                double *vy, double *ax, double *ay, 
                                double *m, double *args, const int bodies)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  double elapse = args[0];
  double space = args[1];
  double radius = args[2];
  double COLLISION_RATIO = args[3];
  if (idx < bodies){
    vx[idx] += ax[idx] * elapse;
    vy[idx] += ay[idx] * elapse;

    bool flag = false;
    if (x[idx] <= radius) {
      flag = true;
      x[idx] = radius + radius * COLLISION_RATIO;
      vx[idx] = -vx[idx];
    } else if (x[idx] >= space - radius) {
      flag = true;
      x[idx] = space - radius - radius * COLLISION_RATIO;
      vx[idx] = -vx[idx];
    }

    if (y[idx] <= radius) {
      flag = true;
      y[idx] = radius + radius * COLLISION_RATIO;
      vy[idx] = -vy[idx];
    } else if (y[idx] >= space - radius) {
      flag = true;
      y[idx] = space - radius - radius * COLLISION_RATIO;
      vy[idx] = -vy[idx];
    }
    if (flag) {
      ax[idx] = 0;
      ay[idx] = 0;
    }

    x[idx] += vx[idx] * elapse;
    y[idx] += vy[idx] * elapse;

    flag = false;
    if (x[idx] <= radius) {
      flag = true;
      x[idx] = radius + radius * COLLISION_RATIO;
      vx[idx] = -vx[idx];
    } else if (x[idx] >= space - radius) {
      flag = true;
      x[idx] = space - radius - radius * COLLISION_RATIO;
      vx[idx] = -vx[idx];
    }

    if (y[idx] <= radius) {
      flag = true;
      y[idx] = radius + radius * COLLISION_RATIO;
      vy[idx] = -vy[idx];
    } else if (y[idx] >= space - radius) {
      flag = true;
      y[idx] = space - radius - radius * COLLISION_RATIO;
      vy[idx] = -vy[idx];
    }
    if (flag) {
      ax[idx] = 0;
      ay[idx] = 0;
    }
    
  }
}

int main(int argc, char **argv) {

    // TODO: 
    // - change all attributes into dynamic arrays then send with hipMalloc
    // - rewrite all functions in device functions
    // - the first nest for loop might not be paralleled as for preventing data racing
    // - so we may only parallel the second for loop
    // 
    // - to prevent data race, we need to investigate co-collision case

  UNUSED(argc, argv);
  static float gravity = 100;
  static float space = 800;
  static float radius = 5;
  static const int bodies = 20;
  static float elapse = 0.1;
  static ImVec4 color = ImVec4(1.0f, 1.0f, 0.4f, 1.0f);
  static float max_mass = 50;
  static const int THREAD_NUMS_PER_BLOCK = 24;

  static float current_space = space;
  static float current_max_mass = max_mass;
  static int current_bodies = bodies;
  BodyPool pool(static_cast<size_t>(bodies), space, max_mass);
  graphic::GraphicContext context{"Assignment 3 CUDA version"};
  context.run([&](graphic::GraphicContext *context [[maybe_unused]],
                  SDL_Window *) {
    auto io = ImGui::GetIO();
    ImGui::SetNextWindowPos(ImVec2(0.0f, 0.0f));
    ImGui::SetNextWindowSize(io.DisplaySize);
    ImGui::Begin("Assignment 3 CUDA version", nullptr,
                 ImGuiWindowFlags_NoMove | ImGuiWindowFlags_NoCollapse |
                     ImGuiWindowFlags_NoTitleBar | ImGuiWindowFlags_NoResize);
    ImDrawList *draw_list = ImGui::GetWindowDrawList();
    ImGui::Text("Application average %.3f ms/frame (%.1f FPS)",
                1000.0f / ImGui::GetIO().Framerate, ImGui::GetIO().Framerate);
    ImGui::DragFloat("Space", &current_space, 10, 200, 1600, "%f");
    ImGui::DragFloat("Gravity", &gravity, 0.5, 0, 1000, "%f");
    ImGui::DragFloat("Radius", &radius, 0.5, 2, 20, "%f");
    ImGui::DragInt("Bodies", &current_bodies, 1, 2, 100, "%d");
    ImGui::DragFloat("Elapse", &elapse, 0.1, 0.001, 10, "%f");
    ImGui::DragFloat("Max Mass", &current_max_mass, 0.5, 5, 100, "%f");
    ImGui::ColorEdit4("Color", &color.x);
    if (current_space != space || current_bodies != bodies ||
        current_max_mass != max_mass) {
      space = current_space;
      // bodies = current_bodies;
      max_mass = current_max_mass;
      pool = BodyPool{static_cast<size_t>(bodies), space, max_mass};
    }
    {
      const ImVec2 p = ImGui::GetCursorScreenPos();

      // pool.update_for_tick(elapse, gravity, space, radius);

      // allocate host memory space
      // 7 attributes, radius, gravity, elapse
      // cuda dont support STL, we only have thrust vector
      double *x =  new double[bodies];
      double *y =  new double[bodies];
      double *vx = new double[bodies];
      double *vy = new double[bodies];
      double *ax = new double[bodies];
      double *ay = new double[bodies];
      double *m = new double[bodies];
      double *args = new double[4]; // stores elapse, space, radius, collision ratio

      // allocate device memory space
      double *d_x, *d_y, *d_vx, *d_vy, *d_ax, *d_ay, *d_m, *d_args;
      hipMalloc(&d_x, bodies*sizeof(double)); //d_x means device_x_coordinate
      hipMalloc(&d_y, bodies*sizeof(double));
      hipMalloc(&d_vx, bodies*sizeof(double));
      hipMalloc(&d_vy, bodies*sizeof(double));
      hipMalloc(&d_ax, bodies*sizeof(double));
      hipMalloc(&d_ay, bodies*sizeof(double));
      hipMalloc(&d_m, bodies*sizeof(double));
      hipMalloc(&d_args, 4*sizeof(double));

      // initialize host data
      pool.ax.assign(pool.size(), 0);
      pool.ay.assign(pool.size(), 0);
      for (size_t i = 0; i < pool.size(); ++i) {
        for (size_t j = i + 1; j < pool.size(); ++j) {
          // update acceleration
          pool.check_and_update(pool.get_body(i), pool.get_body(j), radius, gravity);
        }
      }
      for (int i = 0; i < bodies; i++){
        x[i] = pool.x[i];
        y[i] = pool.y[i];
        vx[i] = pool.vx[i];
        vy[i] = pool.vy[i];
        ax[i] = pool.ax[i];
        ay[i] = pool.ay[i];
        m[i] = pool.m[i];
      }
      args[0] = elapse;
      args[1] = space;
      args[2] = radius;
      args[3] = pool.COLLISION_RATIO;

      // copy host data to device data

      hipMemcpy(d_x, x, bodies*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_y, y, bodies*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_vx, vx, bodies*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_vy, vy, bodies*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_ax, ax, bodies*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_ay, ay, bodies*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_m, m, bodies*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_args, args, 4*sizeof(double), hipMemcpyHostToDevice);

      // call kernel
      cudaCheckBodies<<<(bodies + THREAD_NUMS_PER_BLOCK - 1) / THREAD_NUMS_PER_BLOCK, THREAD_NUMS_PER_BLOCK>>>
      (x, y, vx, vy, ax, ay, m, args, bodies);

      // copy data back to host
      hipMemcpy(x, d_x, bodies*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(y, d_y, bodies*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(vx, d_vx, bodies*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(vy, d_vy, bodies*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(ax, d_ax, bodies*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(ay, d_ay, bodies*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(m, d_m, bodies*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(args, d_args, 4*sizeof(double), hipMemcpyDeviceToHost);

      for (int i = 0; i < bodies; i++){
        pool.x[i] = x[i];
        pool.y[i] = y[i];
        pool.vx[i] = vx[i];
        pool.vy[i] = vy[i];
        pool.ax[i] = ax[i];
        pool.ay[i] = ay[i];
        pool.m[i] = m[i];
      }
      

      // for (size_t i = 0; i < pool.size(); ++i) {
      //   // update position and velocity according to acceleration
      //   pool.get_body(i).update_for_tick(elapse, space, radius);
      // }

      // display only needs x and y
      for (size_t i = 0; i < pool.size(); ++i) {
        auto body = pool.get_body(i);
        auto x = p.x + static_cast<float>(body.get_x());
        auto y = p.y + static_cast<float>(body.get_y());
        draw_list->AddCircleFilled(ImVec2(x, y), radius, ImColor{color});
      }
    }
    ImGui::End();
  });
}
